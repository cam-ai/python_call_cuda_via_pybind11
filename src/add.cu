#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include ""    
#include "stdio.h"
#include "add.cuh"
//CUDA核函数
__global__ void addKernel(double* c, const double* a, const double* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
// 向量相加
void vectorAdd(double c[], double a[], double b[],int size)
{
	double* dev_a = 0;
	double* dev_b = 0;
	double* dev_c = 0;

	//printf("GPU A: \n");
	//for (int i = 0; i < size; i++) {
	//    printf("%.2f ", a[i]);
	//}
	//printf("\n");

	//printf("GPU B: \n");
	//for (int i = 0; i < size; i++) {
	//    printf("%.2f ", b[i]);
	//}
	//printf("\n");
	
	// 在GPU中为变量dev_a、dev_b、dev_c分配内存空间.  
	hipMalloc((void**)&dev_c, size * sizeof(double));
	hipMalloc((void**)&dev_a, size * sizeof(double));
	hipMalloc((void**)&dev_b, size * sizeof(double));

	// 从主机内存复制数据到GPU内存中.  
	hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);

	// 启动GPU内核函数  
	addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

	// 采用hipDeviceSynchronize等待GPU内核函数执行完成并且返回遇到的任何错误信息  
	hipDeviceSynchronize();

	// 从GPU内存中复制数据到主机内存中  
	hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);

	//printf("GPU C: \n");
	//for (int i = 0; i < size; i++) {
	//    printf("%.2f ", c[i]);
	//}
	//printf("\n");
	
	//释放设备中变量所占内存  
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return ;
}
